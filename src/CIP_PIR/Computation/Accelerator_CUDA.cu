#include "hip/hip_runtime.h"


#include <hip/hip_runtime.h>
#include <stdio.h>
#include "Accelerator_CUDA.h"

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void vector_xor(unsigned char* dest, unsigned char* data) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Handling arbitrary vector size
    if (tid < BLOCKSIZE) {
        dest[tid] ^= data[tid];
    }
}

// split it in two parts
// first xor all per block
// then xor the different dest results with eachother
__global__ void vector_xor2(unsigned char* db, unsigned char* dest, unsigned char* data, std::size_t groups) {
    int destS[ELEMENTS_PER_THREAD];

    int bid = blockIdx.x;
    unsigned char* dataPtr = data + bid * groups;

    int threadOffset = threadIdx.x * ELEMENTS_PER_THREAD * 4;

    std::size_t q = static_cast<std::size_t> (dataPtr[0]) * BLOCKSIZE;

    // for first element no xor necessary, since xor with 0 is useless
    for (int i = 0; i < ELEMENTS_PER_THREAD; ++i) {
        destS[i] = *((int*)(db + q + 4 * i + threadOffset));
    }

    // loop over rest of the query
    for (int g = 1; g < groups; ++g) {
        q = static_cast<std::size_t> (dataPtr[g]) * BLOCKSIZE + g * preOffsetZ;
        for (int i = 0; i < ELEMENTS_PER_THREAD; ++i) {
            destS[i] ^= *((int*)(db + q + 4 * i + threadOffset));
        }
    }

    int* destOutPtr = (int*)(dest + bid * BLOCKSIZE);

    // copy from shared to global
    for (int i = 0; i < ELEMENTS_PER_THREAD; ++i) {
        destOutPtr[i + threadOffset / 4] = destS[i];
    }
}
/*
size_t Accelerator_CUDA::getFreeMemory(){
    size_t free, total;
    hipSetDevice( GPU_ID );
    hipMemGetInfo( &free, &total );
    return free;
}*/

Accelerator_CUDA::Accelerator_CUDA(byte* data, size_t size) {
    HANDLE_ERROR(hipMalloc((void**)&db, sizeof(byte) * size));
    HANDLE_ERROR(hipMemcpy(db, data, sizeof(byte) * size, hipMemcpyHostToDevice));
}

Accelerator_CUDA::~Accelerator_CUDA(){
    HANDLE_ERROR(hipFree(db));
}

void Accelerator_CUDA::compute(uint8_t* dst, uint8_t* data_in, const uint64_t groups, int count) {
    byte* dest;
    byte* data;

    HANDLE_ERROR(hipMalloc((byte**)&dest, BLOCKSIZE * count));
    //hipMalloc((void**)&data, sizeof(char) * BLOCKSIZE * count);
    HANDLE_ERROR(hipMalloc((byte**)&data, groups * count));

    HANDLE_ERROR(hipMemset(dest, 0, BLOCKSIZE * count));
    // copy query to GPU
    HANDLE_ERROR(hipMemcpy(data, data_in, groups * count, hipMemcpyHostToDevice));

    auto data_ptr = data;

    // loop over all seeds
    int j;
    for (j = 0; j < count; j += CUDA_BLOCKS) {
        if(j+CUDA_BLOCKS > count) {
            vector_xor2 << < count - j, CUDA_THREADS >> > (db, (dest + j * BLOCKSIZE), data_ptr, groups);
        }
        else
            vector_xor2 << < CUDA_BLOCKS, CUDA_THREADS >> > (db, (dest + j * BLOCKSIZE), data_ptr, groups);
        data_ptr += CUDA_BLOCKS;
    }

    // handle last call; prohibit launching too many blocks
    //vector_xor2 << < count - CUDA_BLOCKS * j, CUDA_THREADS >> > (db, (dest + j * BLOCKSIZE), data_ptr, groups);

    HANDLE_ERROR(hipMemcpy(dst, dest, BLOCKSIZE * count, hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipFree(dest));
    HANDLE_ERROR(hipFree(data));
}

void Accelerator_CUDA::allocateDB(uint8_t* db, uint64_t groupsForServer, uint64_t groupbytes){
    HANDLE_ERROR(hipHostMalloc( (uint8_t**) &db, sizeof(uint8_t)*groupsForServer * groupbytes) );
}

void Accelerator_CUDA::xorFullBlocks(uint8_t *dest, uint8_t *data){
    uint8_t* a;
    uint8_t* b;

    hipMalloc((void**)&a, sizeof(char) * BLOCKSIZE);
    hipMalloc((void**)&b, sizeof(char) * BLOCKSIZE);

    hipMemset(dest, 0, sizeof(char) * BLOCKSIZE);
    hipMemcpy(b, data, BLOCKSIZE, hipMemcpyHostToDevice);

    vector_xor << < CUDA_BLOCKS, CUDA_THREADS >> > (a, b);

    hipMemcpy(dest, a, BLOCKSIZE, hipMemcpyDeviceToHost);

    hipFree(a);
    hipFree(b);
}

void Accelerator_CUDA::fastXOR(uint8_t *dest, uint8_t *data, const uint8_t *query, const uint64_t groups) {
    uint8_t *destC;
    uint8_t *queryC;

    hipMalloc((void**)&destC, sizeof(char) * BLOCKSIZE);
    hipMalloc((void**)&queryC, sizeof(char) * BLOCKSIZE);

    hipMemcpy(destC, dest, BLOCKSIZE, hipMemcpyHostToDevice);
    hipMemcpy(queryC, query, BLOCKSIZE, hipMemcpyHostToDevice);

    vector_xor2 << < 1, CUDA_THREADS >> > (db, destC, queryC, groups);
    hipMemcpy(dest, destC, BLOCKSIZE, hipMemcpyDeviceToHost);

    hipFree(destC);
    hipFree(queryC);
}